#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "CudaContext.hh"
#include "CudaDevice.hh"
#include "utils.hh"

namespace CUDA {

    Context::Context() {
    }

    Context::~Context() {
    }

    int Context::getNumDevices() const {
	int numDevices;
	hipError_t cec;
	cec = hipGetDeviceCount( &numDevices ); CHKCERR( cec );
	return numDevices;
    }

}
